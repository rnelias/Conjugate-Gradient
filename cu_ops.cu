#include "hip/hip_runtime.h"
/* cu_ops.cu - CUDA helper functions
 *
 * CSC 564 - CG Assignment Series
 *
 * Author: Dean Pucsek
 * Date: 2 March 2012
 *
 */

#include "cu_ops.cuh"
#include "cuPrintf.cu"

#define BLOCK_SIZE 4

/* ---------- Matrix-Vector Operations ---------- */

__global__ void cgMVMult(Matrix *mat_A, Vector *vec_b, Vector *vec_c)
{
    int mx_idx = threadIdx.x + (blockDim.x * threadIdx.y);
    int ve_idx = threadIdx.x;
    unsigned int s = 0;
    double temp_res, row_res;
    __shared__ double temp_mat[BLOCK_SIZE][BLOCK_SIZE];

    temp_res = mat_A->values[mx_idx] * vec_b->values[ve_idx];
    temp_mat[threadIdx.y][threadIdx.x] = temp_res;
    __syncthreads();

    for(s = 1; s < blockDim.x; s *= 2)
    {
        if(ve_idx % (2*s) == 0)
        {
            temp_mat[threadIdx.y][threadIdx.x] += temp_mat[threadIdx.y][threadIdx.x + s];
        }
    }
    
    row_res = temp_mat[threadIdx.y][0];
    if(ve_idx == 0)
        vec_c->values[threadIdx.y] = row_res;
}

__global__ void cgReduce(double *dp, int dp_size, double *dp_final)
{
    unsigned int s;
    int tid = threadIdx.x;

    for(s = 1; s < blockDim.x; s *= 2)
    {
        if(tid % (2*s) == 0)
        {
            dp[tid] += dp[tid + s];
        }

        __syncthreads();
    }

    *dp_final = dp[0];
}

__global__ void cgDotProduct(Vector *vec_a, Vector *vec_b, double *dp_final)
{
    unsigned int s;
    int i = threadIdx.x;
    __shared__ double dp[BLOCK_SIZE];

    dp[i] = vec_a->values[i] * vec_b->values[i];
    __syncthreads();

    for(s = 1; s < blockDim.x; s *= 2)
    {
        if(i % (2*s) == 0)
        {
            dp[i] += dp[i + s];
        }

        __syncthreads();
    }

    *dp_final = dp[0];
}

__global__ void cgSVMult(double *sca, Vector *vec_a, Vector *vec_b)
{
    int i = threadIdx.x;
    vec_b->values[i] = (*sca) * vec_a->values[i];
}

__global__ void cgVecSub(Vector *vec_a, Vector *vec_b, Vector *vec_c)
{
    int i = threadIdx.x;
    vec_c->values[i] = vec_a->values[i] - vec_b->values[i];
}

__global__ void cgVecAdd(Vector *vec_a, Vector *vec_b, Vector *vec_c)
{
    int i = threadIdx.x;
    vec_c->values[i] = vec_a->values[i] + vec_b->values[i];
}

/* ---------- Device Memory Management ---------- */
void cgDeepCopy(Vector *pdvec_a, Vector *pdvec_b)
{
}

/* ---------- Copying between host and device ---------- */

int cgCopyMatrix(Matrix *h_m, Matrix **d_m)
{
    hipError_t cudaResult;
    Matrix h_temp;
    double *d_values;
    int *d_column_indices, *d_row_pointers;

    /* Allocate space and copy values */
    cudaResult = hipMalloc(&d_values, sizeof(double) * h_m->nnz);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    cudaResult = hipMemcpy(d_values, h_m->values, sizeof(double) * h_m->nnz, hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    /* Allocate space and copy column indices */
    cudaResult = hipMalloc(&d_column_indices, sizeof(int) * h_m->nnz);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    cudaResult = hipMemcpy(d_column_indices, h_m->column_indices, sizeof(int) * h_m->nnz, hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    /* Allocate space and copy row pointers */
    cudaResult = hipMalloc(&d_row_pointers, sizeof(int) * (h_m->size + 1));
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    cudaResult = hipMemcpy(d_row_pointers, h_m->row_pointers, sizeof(int) * (h_m->size + 1), hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }
    
    /* Allocate space on device for matrix structure */
    cudaResult = hipMalloc(d_m, sizeof(Matrix));
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    h_temp.size = h_m->size;
    h_temp.nnz = h_m->nnz;
    h_temp.values = d_values;
    h_temp.column_indices = d_column_indices;
    h_temp.row_pointers = d_row_pointers;

    cudaResult = hipMemcpy(*d_m, &h_temp, sizeof(Matrix), hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    return 0;
}

int cgCopyVectorToHost(Vector *d_v, Vector **pph_v)
{
    hipError_t cudaResult;
    double *h_values;

    if(*pph_v == NULL)
    {
        *pph_v = (Vector *)calloc(1, sizeof(Vector));
    }

    cudaResult = hipMemcpy(*pph_v, d_v, sizeof(Vector), hipMemcpyDeviceToHost);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error cgCopyVectorToHost-1: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    h_values = (double *)calloc((*pph_v)->size, sizeof(double));
    if(!h_values)
    {
        fprintf(stderr, "Error: failed to allocate spcae for host values\n");
        return -1;
    }

    cudaResult = hipMemcpy(h_values, (*pph_v)->values, (*pph_v)->size * sizeof(double), hipMemcpyDeviceToHost);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error cgCopyVectorToHost-2: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    (*pph_v)->values = h_values;

    return 0;
}

int cgCopyVector(Vector *h_v, Vector **d_v)
{
    hipError_t cudaResult;
    Vector h_temp;
    double *d_values;

    /* Allocate space and copy values */
    cudaResult = hipMalloc(&d_values, sizeof(double) * h_v->size);
    if(cudaResult != hipSuccess) 
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    cudaResult = hipMemcpy(d_values, h_v->values, h_v->size * sizeof(double), hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess) 
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    /* Allocate space on the device for the vector structure */
    h_temp.values = d_values;
    h_temp.size = h_v->size;

    cudaResult = hipMalloc(d_v, sizeof(Vector));
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    cudaResult = hipMemcpy(*d_v, &h_temp, sizeof(Vector), hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    return 0;
}

int cgCloneVector(Vector *phvec_a, Vector **pphvec_b)
{
    return 1;
}

/* ---------- Helper Kernels ---------- */

/* Note: "Vector *v" produces a warning during compilation about NVCC not being able to deduce where the pointer points.  This 
 * is because (1) cards with compute capability 1.x have separate address spaces for global and shared memory and (2) the pointer
 * can actually point to either.  NVCC assumes global memory which (in this case) is correct.
 */
__global__ void cgPrintVector(Vector *v)
{
    int i = threadIdx.x;

    cuPrintf("Thread %d has value %f\n", i, v->values[i]);
}
