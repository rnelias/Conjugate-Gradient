#include "hip/hip_runtime.h"
/* cu_ops.cu - CUDA helper functions
 *
 * CSC 564 - CG Assignment Series
 *
 * Author: Dean Pucsek
 * Date: 2 March 2012
 *
 */

#include "cu_ops.cuh"
#include "cuPrintf.cu"

/* ---------- Matrix-Vector Operations ---------- */

__device__ void cgMVMult(Matrix mat_A, Vector vec_b, Vector *vec_c)
{
    int mx_idx = threadIdx.x + (blockDim.x * threadIdx.y);
    int ve_idx = threadIdx.x;
    unsigned int s = 0;
    double temp_res, row_res;
    __shared__ double temp_mat[8][8];

    temp_res = mat_A.values[mx_idx] * vec_b.values[ve_idx];
    temp_mat[threadIdx.y][threadIdx.x] = temp_res;
    __syncthreads();

    for(s = 1; s < blockDim.x; s *= 2)
    {
        if(ve_idx % (2*s) == 0)
        {
            temp_mat[threadIdx.y][threadIdx.x] += temp_mat[threadIdx.y][threadIdx.x + s];
        }
    }
    
    row_res = temp_mat[threadIdx.y][0];
    if(ve_idx == 0)
        vec_c->values[threadIdx.y] = row_res;
}

__device__ void cgReduce(double *dp, int dp_size, double *dp_final)
{
    unsigned int s;
    int tid = threadIdx.x;

    for(s = 1; s < blockDim.x; s *= 2)
    {
        if(tid % (2*s) == 0)
        {
            dp[tid] += dp[tid + s];
        }

        __syncthreads();
    }

    *dp_final = dp[0];
}

__device__ void cgDotProduct(Vector vec_a, Vector vec_b, double *dp)
{
    int i = threadIdx.x;
    dp[i] = vec_a.values[i] * vec_b.values[i];
}

__device__ void cgSVMult(int sca, Vector vec_a, Vector *vec_b)
{
    int i = threadIdx.x;
    vec_b->values[i] = sca * vec_a.values[i];
}

__device__ void cgVecSub(Vector vec_a, Vector vec_b, Vector *vec_c)
{
    int i = threadIdx.x;
    vec_c->values[i] = vec_a.values[i] - vec_b.values[i];
}

__device__ void cgVecAdd(Vector vec_a, Vector vec_b, Vector *vec_c)
{
    int i = threadIdx.x;
    vec_c->values[i] = vec_a.values[i] + vec_b.values[i];
}

/* ---------- Device Memory Management ---------- */
Vector **cgDeviceAllocateVectorArray(Vector **hv_array, int arr_size, Vector *hv_template)
{
    hipError_t cudaResult;
    Vector *dv_template, *dv_array;
    int idx = 0;

    for(idx = 0; idx < arr_size; idx++)
    {
        printf("-> Setting dv_template\n");
        //cgCopyVector(hv_template, &dv_template);
        hipMalloc(&dv_template, sizeof(Vector));
        hipMemcpy(dv_template, hv_template, sizeof(Vector), hipMemcpyHostToDevice);
        hv_array[idx] = dv_template;
        printf("-> Finished with dv_template\n");
    }

    /* Allocate and copy to device */
    cudaResult = hipMalloc(&dv_array, arr_size * sizeof(Vector *));
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error cgDeviceAllocateVectorArray-1: %s\n", hipGetErrorString(cudaResult));
        return NULL;
    }

    printf("dv_array: %p\n", dv_array);
    printf("hv_array: %p\n", hv_array);
    for(idx = 0; idx < arr_size; idx++)
    {
        printf("hv_array[%d]: %p\n", idx, hv_array[idx]);
    }

    cudaResult = hipMemcpy(dv_array, hv_array, arr_size * sizeof(Vector *), hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error cgDeviceAllocateVectorArray-2: %s\n", hipGetErrorString(cudaResult));
        return NULL;
    }

    Vector *pdv_array;
    cudaResult = hipMalloc(&pdv_array, sizeof(Vector *));
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error cgDeviceAllocateVectorArray-3: %s\n", hipGetErrorString(cudaResult));
        return NULL;
    }

    cudaResult = hipMemcpy(pdv_array, &dv_array, sizeof(Vector *), hipMemcpyDeviceToDevice);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error cgDeviceAllocateVectorArray-4: %s\n", hipGetErrorString(cudaResult));
        return NULL;
    }

    return (Vector **)pdv_array;
}

__device__ void cgDeepCopy(Vector vec_a, Vector *pvec_b)
{
    pvec_b->size = vec_a.size;
    if(threadIdx.x == 0)
        pvec_b->values[threadIdx.y] = vec_a.values[threadIdx.y];    
}

/* ---------- Copying between host and device ---------- */

int cgCopyMatrix(Matrix *h_m, Matrix **d_m)
{
    hipError_t cudaResult;
    Matrix h_temp;
    double *d_values;
    int *d_column_indices, *d_row_pointers;

    /* Allocate space and copy values */
    cudaResult = hipMalloc(&d_values, sizeof(double) * h_m->nnz);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    cudaResult = hipMemcpy(d_values, h_m->values, sizeof(double) * h_m->nnz, hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    /* Allocate space and copy column indices */
    cudaResult = hipMalloc(&d_column_indices, sizeof(int) * h_m->nnz);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    cudaResult = hipMemcpy(d_column_indices, h_m->column_indices, sizeof(int) * h_m->nnz, hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    /* Allocate space and copy row pointers */
    cudaResult = hipMalloc(&d_row_pointers, sizeof(int) * (h_m->size + 1));
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    cudaResult = hipMemcpy(d_row_pointers, h_m->row_pointers, sizeof(int) * (h_m->size + 1), hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }
    
    /* Allocate space on device for matrix structure */
    cudaResult = hipMalloc(d_m, sizeof(Matrix));
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    h_temp.size = h_m->size;
    h_temp.nnz = h_m->nnz;
    h_temp.values = d_values;
    h_temp.column_indices = d_column_indices;
    h_temp.row_pointers = d_row_pointers;

    cudaResult = hipMemcpy(*d_m, &h_temp, sizeof(Matrix), hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    return 0;
}

int cgCopyVectorToHost(Vector *d_v, Vector *h_v)
{
    hipError_t cudaResult;
    double *h_values;
    
    printf("cgCopyVectorToHost: h_v -> %p\n", h_v);
    printf("cgCopyVectorToHost: d_v -> %p\n", d_v);

    cudaResult = hipMemcpy(h_v, d_v, sizeof(Vector), hipMemcpyDeviceToHost);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error cgCopyVectorToHost-1: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    h_values = (double *)calloc(h_v->size, sizeof(double));
    if(!h_values)
    {
        fprintf(stderr, "Error: failed to allocate spcae for host values\n");
        return -1;
    }

    printf("h_v->values: %p\n", h_v->values);

    cudaResult = hipMemcpy(h_values, h_v->values, h_v->size * sizeof(double), hipMemcpyDeviceToHost);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error cgCopyVectorToHost-2: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    h_v->values = h_values;

    return 0;
}

int cgCopyVector(Vector *h_v, Vector **d_v)
{
    hipError_t cudaResult;
    Vector h_temp;
    double *d_values;

    /* Allocate space and copy values */
    cudaResult = hipMalloc(&d_values, sizeof(double) * h_v->size);
    if(cudaResult != hipSuccess) 
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    cudaResult = hipMemcpy(d_values, h_v->values, h_v->size * sizeof(double), hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess) 
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    /* Allocate space on the device for the vector structure */
    h_temp.values = d_values;
    h_temp.size = h_v->size;
    printf("h_v->size: %d\n", h_v->size);

    cudaResult = hipMalloc(d_v, sizeof(Vector));
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    cudaResult = hipMemcpy(*d_v, &h_temp, sizeof(Vector), hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaResult));
        return -1;
    }

    printf("cgCopyVector: %p\n", *d_v);
    return 0;
}


/* ---------- Helper Kernels ---------- */

/* Note: "Vector *v" produces a warning during compilation about NVCC not being able to deduce where the pointer points.  This 
 * is because (1) cards with compute capability 1.x have separate address spaces for global and shared memory and (2) the pointer
 * can actually point to either.  NVCC assumes global memory which (in this case) is correct.
 */
__global__ void cgPrintVector(Vector *v)
{
    int i = threadIdx.x;

    cuPrintf("Thread %d has value %f\n", i, v->values[i]);
}
