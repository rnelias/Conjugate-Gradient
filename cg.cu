#include "hip/hip_runtime.h"
/* cg.c - Conjugate Gradient, in all it's glory!
 *
 * CSC 564 - CG Assignment Series
 *
 * Author: Dean Pucsek
 * Date: 8 March 2012
 *
 */

#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "mv_types.cuh"
#include "cu_ops.cu"
#include "reader.cu"

#ifndef TRUE
#define TRUE 1
#define FALSE 0
#endif

/* Controls the number of times the CG is 
 * executed (handy for collecting data) 
 */
#define MAX_EXECUTION_COUNT 1

/* Controls the maximum number of CUDA threads
 * used for the CG computation.
 */
#define MAX_THREAD_COUNT_SHIFT

/* The size (height and width) of each thread
 * block.  Currently this must be a power of two.
 */
#define BLOCK_SIZE 8

/* ---------- Conjugate Gradient ---------- */
__global__ void cgConjGrad(int, Matrix *, Vector *, Vector *, Vector **);
__global__ void cgTestMVOps(Matrix *, Vector *);

int main(int argc, char **argv)
{
  const char *input_file = NULL;
  int max_iterations = -1;
  int exec_count = 0, i = 0;

  /* Process command arguments */
  if(argc < 3) {
    fprintf(stderr, "Usage: %s <input-data> <max-iterations> [suppress-output]\n", argv[0]);
    return -1;
  }

  input_file = argv[1];
  max_iterations = (int)strtol(argv[2], NULL, 10);

  /* Figure out the number of blocks and threads per block */
  int numBlocks = 1;
  dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

  hipEvent_t start, stop;
  float time;

  /* CUDA printf() initialization */
  cudaPrintfInit();

  /* Initialize CUDA events */
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Initialize matrix A and vector b */
  Matrix *dm_A, hm_A;
  Vector *dv_b, hv_b;
  Vector *dv_x, hv_x;

  /* Read input */
  read_input_file(input_file, &hm_A, &hv_b);
  
  /* Send data to GPU */
  cgCopyMatrix(&hm_A, &dm_A);
  cgCopyVector(&hv_b, &dv_b);

  if(hipMalloc(&dv_x, sizeof(Vector)) != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate space for dv_x\n");
  }
  
  /* Test MV Ops */
  //cgTestMVOps<<<numBlocks, threadsPerBlock>>>(dm_A, dv_b);
  //hipDeviceSynchronize();
  //cudaPrintfDisplay(stdout, true);

  /* Allocate the various vectors needed for the CG method */
  Vector *hv_array[7];
  Vector **dv_array = cgDeviceAllocateVectorArray(hv_array, 7, &hv_b);
  
  /* Compute CG */
  printf("\tThreads\tTime\n");
  for(exec_count = 0; exec_count < MAX_EXECUTION_COUNT; exec_count++) 
  {
      hipEventRecord(start, 0);
      cgConjGrad<<<numBlocks, threadsPerBlock>>>(max_iterations, dm_A, dv_b, dv_x, dv_array);
      cudaPrintfDisplay(stdout, false);
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      
      hipEventElapsedTime(&time, start, stop);

      cgCopyVectorToHost(dv_x, &hv_x);
      printf("hv_x:\n");
      printf("\tsize: %d\n", hv_x.size);
      printf("\tvalues: %p\n", hv_x.values);
      if(hv_x.values == NULL)
      {
          printf("\thv_x.values: NULL\n");
      }
      else
      {
          for(i = 0; i < hv_x.size; i++)
              printf("\thv_x.values[%d]: %f\n", i, hv_x.values[i]);
      }
      
      printf("%04d\t%d\t%f\n", exec_count, threadsPerBlock.x * threadsPerBlock.y, time);
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);
  cudaPrintfEnd();

  return 0;
}

/* ---------- Conjugate Gradient ---------- */

__global__ void cgConjGrad(int max_iterations, Matrix *pmat_A, Vector *pvec_b, Vector *pvec_x, Vector **pvec_array)
{
    __shared__ double dp_shared[BLOCK_SIZE];
    double dp_res_1, dp_res_2, sca_alpha, sca_beta;
    __shared__ int k;

    Matrix mat_A;
    Vector vec_b;

    Vector *pvec_r, *pvec_s, *pvec_p;
    Vector *psv_res, *pvx_old, *pvx_new, *pvec_pr;
    
    mat_A = *pmat_A;
    vec_b = *pvec_b;

    k = 0;

    pvec_r = pvec_array[0];
    pvec_s = pvec_array[1];
    pvec_p = pvec_array[2];
    
    psv_res = pvec_array[3];
    pvx_old = pvec_array[4];
    pvx_new = pvec_array[5];
    pvec_pr = pvec_array[6];

    cuPrintf("1 pvec_r: %p\n", pvec_r);
    cuPrintf("1 pvec_r->size: %d\n", pvec_r->size);
    
    cgDeepCopy(vec_b, pvec_r);
    cgDeepCopy(*pvec_r, pvec_p);

    /*
    cgDeepCopy(vec_b, &vec_s);
    cgDeepCopy(vec_b, &sv_res);
    cgDeepCopy(vec_b, &vx_old);
    cgDeepCopy(vec_b, &vx_new);
    cgDeepCopy(vec_b, &vec_prev_r);
    */
    
    cuPrintf("2 pvec_r->size: %d\n", pvec_r->size);
    cuPrintf("2 pvec_r->values: %p\n", pvec_r->values);
    cuPrintf("2 pvec_r->values[%d]: %f\n", threadIdx.y, pvec_r->values[threadIdx.y]);
    
    while(TRUE)
    {
        cgMVMult(mat_A, *pvec_p, pvec_s);
        __syncthreads();

        cgDotProduct(*pvec_r, *pvec_r, dp_shared);
        cgReduce(dp_shared, blockDim.x, &dp_res_1);
        __syncthreads();

        cgDotProduct(*pvec_p, *pvec_s, dp_shared);
        cgReduce(dp_shared, blockDim.x, &dp_res_2);
        __syncthreads();

        sca_alpha = dp_res_1 / dp_res_2;

        cgSVMult(sca_alpha, *pvec_p, psv_res);
        __syncthreads();

        cgDeepCopy(*pvx_new, pvx_old);
        cgVecAdd(*pvx_old, *psv_res, pvx_new);
        __syncthreads();

        cgDeepCopy(*pvec_r, pvec_pr);
        cgSVMult(sca_alpha, *pvec_s, psv_res);
        __syncthreads();

        cgVecSub(*pvec_r, *psv_res, pvec_r);
        __syncthreads();

        if(k == max_iterations)
        {
            break;
        }

        cgDotProduct(*pvec_r, *pvec_r, dp_shared);
        cgReduce(dp_shared, blockDim.x, &dp_res_1);
        __syncthreads();

        cgDotProduct(*pvec_pr, *pvec_pr, dp_shared);
        cgReduce(dp_shared, blockDim.x, &dp_res_2);
        __syncthreads();

        sca_beta = dp_res_1 / dp_res_2;

        cgSVMult(sca_beta, *pvec_p, psv_res);
        __syncthreads();

        cgVecAdd(*pvec_r, *psv_res, pvec_p);
        __syncthreads();
        
        k++;
    }

    pvec_x->size = pvx_new->size;
    pvec_x->values = pvx_new->values;
    //cuPrintf("pvec_x->size: %d\n", pvec_x->size);
    //cuPrintf("pvec_x->values: %p\n", pvec_x->values);
}

/* ---------- Testing Matrix and Vector operations ---------- */

__global__ void cgTestMVOps(Matrix *pmat_A, Vector *pvec_b)
{
    //int i = threadIdx.x;
    Vector vec_b, vec_c;
    __shared__ double dp[5];
    double dp_res;
    Matrix mat_A;

    mat_A = *pmat_A;
    vec_b = *pvec_b;

    cgVecAdd(vec_b, vec_b, &vec_c);
    //cuPrintf("cgVecAdd: vec_c[%d] = %f\n", i, vec_c.values[i]);

    cgVecSub(vec_b, vec_b, &vec_c);
    //cuPrintf("cgVecSub: vec_c[%d] = %f\n", i, vec_c.values[i]);

    cgSVMult(4, vec_b, &vec_c);
    //cuPrintf("cgSVMult: vec_c[%d] = %f\n", i, vec_c.values[i]);

    cgDotProduct(vec_b, vec_b, dp);
    cgReduce(dp, 5, &dp_res);
    //cuPrintf("cgDotProduct+cgReduce: %f\n", dp_res);

    cgMVMult(mat_A, vec_b, &vec_c);
    //if(threadIdx.x == 0)
    //    cuPrintf("cgMVMult: vec_c[%d] = %f\n", threadIdx.y, vec_c.values[threadIdx.y]);
}
